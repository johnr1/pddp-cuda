#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/file_io.h"
#include "../include/matrix.h"
#include "../include/pddp.h"
#include "../include/cuda_helper.h"


int main(int argc, char* argv[]) {
    const double e = 10e-6;
    char *input_file, *output_file;
    if (argc < 2){
        printf("Usage: %s input_file [output_file]\n", argv[0]);
        exit(1);
    } else if(argc == 2) {
        input_file = argv[1];
        output_file = "result.mat";
    }
    else{
        input_file = argv[1];
        output_file = argv[2];
    }
    
    printf("Program started\n");
    fflush(stdout);

    // Host
    Matrix M;
    M.matrix = file_read(input_file, &M.rows, &M.cols);
    printf("File read\n");
    fflush(stdout);
    Matrix x = matrixHostMalloc( M.cols, 1);


    // Device
    Matrix d_M = matrixDeviceMalloc(M.rows, M.cols);
    Matrix d_w = matrixDeviceMalloc(M.rows, 1);
    Matrix d_xNext = matrixDeviceMalloc(M.cols, 1);
    Matrix d_x = matrixDeviceMalloc(M.cols, 1);
    Matrix d_temp = matrixDeviceMalloc(M.cols, 1);
    Matrix d_temp2 = matrixDeviceMalloc(M.cols,1);
    Matrix d_mulTemp = matrixDeviceMalloc(M.rows, M.cols/GRID_X + 1);
    Matrix d_mulTemp2 = matrixDeviceMalloc(M.rows, M.cols/GRID_X/GRID_X + 1);
    

    // Transfer M matrix to device
    hipMemcpy(d_M.matrix, M.matrix, M.cols*M.rows*sizeof(double), hipMemcpyHostToDevice);
    cudaCheckError();
    

    // Kernels which calculates avg weight vector and initializes d_x
    calculateAverageVector<<<M.rows/S_BLOCK_SIZE + 1, S_BLOCK_SIZE>>>(d_M,d_w); //Populates d_w
    cudaCheckError();

    initialize<<<d_x.rows/S_BLOCK_SIZE + 1, S_BLOCK_SIZE>>>(d_x,1);             //Populated d_x
    cudaCheckError();

    // Allocate Mapped varianceNorm value
    double *varianceNorm, *d_varianceNorm; //1 iteration
    hipHostAlloc((void **)&varianceNorm, sizeof(double), hipHostMallocMapped);
    hipHostGetDevicePointer((void **)&d_varianceNorm, varianceNorm, 0);


    printf("Memory allocations finished\n");
    fflush(stdout);

    Matrix tempPointer;
    *varianceNorm = 0;
    do {
        d_temp.rows = M.rows;
        subtractAndMultiply(d_M, d_w, d_x, d_mulTemp, d_mulTemp2, d_temp);

        subtractAndMultiplyTranspose<<<M.cols/S_BLOCK_SIZE + 1, S_BLOCK_SIZE>>>(d_M, d_w, d_temp, d_xNext);

        norm(d_xNext,&d_temp,&d_temp2,d_varianceNorm); //d_temp[0] contains norm value
        divMatrixWithNorm<<<(d_xNext.rows/S_BLOCK_SIZE)+1, S_BLOCK_SIZE>>>(d_temp, d_xNext); //Alters d_xNext
        
        d_temp.rows = M.cols;
        subtractMatrix<<<(d_xNext.rows/S_BLOCK_SIZE)+1, S_BLOCK_SIZE>>>(d_xNext, d_x); //Alters d_x
        norm(d_x, &d_temp, &d_temp2, d_varianceNorm); //makes d_temp[0] the norm value

        tempPointer = d_x; //Jungle pointers
        d_x = d_xNext;
        d_xNext = tempPointer;

        hipDeviceSynchronize();
    } while(*varianceNorm > e);


    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(x.matrix, d_x.matrix, d_x.rows*sizeof(double), hipMemcpyDeviceToHost);
    cudaCheckError();

    print_to_file(x, output_file);//printing to file in order to both check values and print debug info

    hipFree(d_M.matrix);
    cudaCheckError();
    hipFree(d_w.matrix);
    cudaCheckError();
    hipFree(d_x.matrix);
    cudaCheckError();
    hipFree(d_xNext.matrix);
    cudaCheckError();
    free(M.matrix);
    free(x.matrix);

    return 0;
}



