#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/file_io.h"
#include "../include/matrix.h"
#include "../include/pddp.h"
#include "../include/cuda_helper.h"

__global__ void reduce(Matrix in, Matrix out, int limit, double* varianceNorm) {
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    int size = limit < blockDim.x ? limit : blockDim.x;

    if(i > in.rows)
        return;

    out.matrix[i] = in.matrix[i] * in.matrix[i]; // wrong ind mallon
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=1; s < size; s *= 2) {
        if (tid % (2*s) == 0 && tid+s < size) {
            out.matrix[i] += out.matrix[i + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0){
        out.matrix[blockIdx.x] = sqrt(out.matrix[blockIdx.x]);
        *varianceNorm = out.matrix[blockIdx.x];
        //printf("DEBUG: %f \n", out.matrix[blockIdx.x]);
    } 
}


void norm(Matrix x, Matrix temp, double* varianceNorm) {
    int counter = 1;
    int threads = S_BLOCK_SIZE;
    int blockSize = x.rows / S_BLOCK_SIZE + 1;
    reduce<<<blockSize, threads>>>(x, temp, x.rows, varianceNorm); 
    if(blockSize == 1){
        printf("Recursive reductions: %d.\n", counter);
        return;
    }

    do{
        counter++;
        int prevBlock = blockSize;
        blockSize = blockSize/threads + 1;
        reduce<<<blockSize, threads>>>(temp, temp, prevBlock, varianceNorm); 
    } while(blockSize > 1);

    printf("Recursive reductions: %d.\n", counter);
    
    cudaCheckError();
}



int main(int argc, char* argv[]) {
    const double e = 10e-6;
    if (argc < 2){
        printf("Usage: %s filename\n", argv[0]);
        exit(1);
    }
    printf("Program started\n");
    fflush(stdout);

    // Host
    Matrix M;
    M.matrix = file_read(argv[1], &M.rows, &M.cols);
    printf("File read\n");
    fflush(stdout);
    Matrix x = matrixHostMalloc( M.cols, 1);


    // Device
    Matrix d_M = matrixDeviceMalloc(M.rows, M.cols);
    Matrix d_w = matrixDeviceMalloc(M.rows, 1);
    Matrix d_xNext = matrixDeviceMalloc(M.cols, 1);
    Matrix d_x = matrixDeviceMalloc(M.cols, 1);
    Matrix d_temp = matrixDeviceMalloc(M.cols, 1);
    

    // Transfer M matrix to device
    hipMemcpy(d_M.matrix, M.matrix, M.cols*M.rows*sizeof(double), hipMemcpyHostToDevice);
    cudaCheckError();
    

    // Kernels which calculates avg weight vector and initializes d_x
    calculateAverageVector<<<M.rows/S_BLOCK_SIZE + 1, S_BLOCK_SIZE>>>(d_M,d_w); //Populates d_w
    cudaCheckError();

    initialize<<<d_x.rows/S_BLOCK_SIZE + 1, S_BLOCK_SIZE>>>(d_x,1);             //Populated d_x
    cudaCheckError();

    // Allocate Mapped varianceNorm value
    double *varianceNorm, *d_varianceNorm; //1 iteration
    hipHostAlloc((void **)&varianceNorm, sizeof(double), hipHostMallocMapped);
    hipHostGetDevicePointer((void **)&d_varianceNorm, varianceNorm, 0);





    /* EXPIRIMENTING 

    Matrix d_temp2 = matrixDeviceMalloc(M.rows, M.cols);
    norm(d_M,d_temp2,d_varianceNorm);
    hipDeviceSynchronize();
    
    exit(0);

    END */











    printf("Memory allocations finished\n");
    fflush(stdout);

    Matrix tempPointer;
    *varianceNorm = 0;
    do {
        d_temp.rows = M.rows;
        subtractAndMultiply<<<M.rows/S_BLOCK_SIZE + 1, S_BLOCK_SIZE>>>(d_M, d_w, d_x, d_temp);
        subtractAndMultiplyTranspose<<<M.cols/S_BLOCK_SIZE + 1, S_BLOCK_SIZE>>>(d_M, d_w, d_temp, d_xNext);

        norm(d_xNext,d_temp,d_varianceNorm); //d_temp[0] contains norm value
        divMatrixWithNorm<<<(d_xNext.rows/S_BLOCK_SIZE)+1, S_BLOCK_SIZE>>>(d_temp, d_xNext); //Alters d_xNext
        
        d_temp.rows = M.cols;
        subtractMatrix<<<(d_xNext.rows/S_BLOCK_SIZE)+1, S_BLOCK_SIZE>>>(d_xNext, d_x); //Alters d_x
        norm(d_x, d_temp, d_varianceNorm); //makes d_temp[0] the norm value

        tempPointer = d_x; //Jungle pointers
        d_x = d_xNext;
        d_xNext = tempPointer;

        hipDeviceSynchronize();
    } while(*varianceNorm > e);






    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(x.matrix, d_x.matrix, d_x.rows*sizeof(double), hipMemcpyDeviceToHost);
    cudaCheckError();

    print(x);

    hipFree(d_M.matrix);
    cudaCheckError();
    hipFree(d_w.matrix);
    cudaCheckError();
    hipFree(d_x.matrix);
    cudaCheckError();
    hipFree(d_xNext.matrix);
    cudaCheckError();
    free(M.matrix);
    free(x.matrix);

    return 0;
}



